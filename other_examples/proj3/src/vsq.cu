#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <proj3.cuh>

__global__
void vsq(int n, float* d_v) {
    const int n_threads = blockDim.x * gridDim.x;
    const int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = thread_idx; i < n; i += n_threads)
        d_v[i] = d_v[i] * d_v[i];
}
