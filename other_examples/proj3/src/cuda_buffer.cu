#include <hipfft/hipfft.h>
#include <proj3.cuh>

CudaBuffer::CudaBuffer(int n): n(n) {
    hipMalloc((void**)&data, sizeof(float)*n);
}

CudaBuffer::~CudaBuffer() {
    hipFree(data);
}
