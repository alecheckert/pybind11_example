#include <iostream>
#include <hipfft/hipfft.h>
#include <proj3.cuh>

#define THREADS_PER_BLOCK 256

int transvsq(int n, float* v, float* out) {
    int n_thread_blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    CudaBuffer B (n);
    hipError_t exit_code = hipMemcpy(B.data, v, sizeof(float)*n, hipMemcpyHostToDevice);
    vsq<<<n_thread_blocks,THREADS_PER_BLOCK>>>(n, B.data);
    exit_code = hipMemcpy(out, B.data, sizeof(float)*n, hipMemcpyDeviceToHost);
    return (int) exit_code;
}
